#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>

__global__ void matmul(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

extern "C" void launch_matmul(float *A, float *B, float *C, int N) {
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (N + 15) / 16);

    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matmul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU Matrix Multiplication Time: %.4f ms\n", milliseconds);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
